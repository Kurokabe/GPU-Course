#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "MandelbrotMath.h"

#include "DomaineMath_GPU.h"
#include "IndiceTools_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public		 	*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, int n)
    {
    MandelbrotMath mandelbrotMath; // warning syntaxe juste !
    //MandelbrotMath mandelbrotMath();// warning syntaxe fausse !

    // TODO Mandelbrot GPU :
    // entrelacement
    // s -> (i,j) -> (x,y)
    // appeler colorXY

    // Choix a faire:
    //		- Travailler en float pour plus de performance?
    //		- Travailler en double pour pouvoir zoomer plus loin dans le fractale?
    // 		Note : domaineMath.toXY travaille lui toujours en double pour vous permettre les deux possibilites ci-dessus
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

