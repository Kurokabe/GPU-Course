#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "MandelbrotMath.h"

#include "DomaineMath_GPU.h"
#include "IndiceTools_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public		 	*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, int n)
    {
    MandelbrotMath mandelbrotMath; // warning syntaxe juste !
    //MandelbrotMath mandelbrotMath();// warning syntaxe fausse !

    const int TID = Indice2D::tid();
        const int NB_THREAD = Indice2D::nbThread();
        const int WH = w * h;

        int i;
        int j;
        float x;
        float y;
        int s = TID;
        while (s < WH)
    	{
            IndiceTools::toIJ(s, w, &i, &j);
            domaineMath.toXY(i, j, &x, &y);
            mandelbrotMath.colorXY(&ptrDevPixels[s], x, y, n);
	    s += NB_THREAD;
    	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

