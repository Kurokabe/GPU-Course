#include "hip/hip_runtime.h"
#include "Rippling.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rippling(uchar4* ptrDevPixels,uint w, uint h,float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Rippling::Rippling(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "Rippling_Cuda_RGBA_uchar4")
    {
    assert(w == h); // specific rippling

    // Animation
    this->dt = dt;
    this->t = 0; // protected dans Animable
    }

Rippling::~Rippling()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car image pas zoomable
 */
void Rippling::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    // TODO Rippling GPU
    // lancer le kernel avec <<<dg,db>>>
    // le kernel est importer ci-dessus (ligne 19)
    rippling<<<dg, db>>>(ptrDevPixels, w, h, t);
    }

/**
 * Override
 * Call periodicly by the API
 */
void Rippling::animationStep()
    {
    t += dt;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

