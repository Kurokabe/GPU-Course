#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"

#include "RipplingMath.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, uint w, uint h, float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    // TODO instacier RipplingMath

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    // TODO Rippling GPU  pattern entrelacement
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

