#include "hip/hip_runtime.h"
#include "Raytracing.h"

#include <iostream>

#include "Device.h"
#include <assert.h>
#include "SphereCreator.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/


/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres);
__global__ void kernelRaytacingSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres);
__global__ void kernelRaytacingCM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSpheres);

__host__ void uploadToCM(Sphere* ptrTabSpheres, int nbSpheres);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur		*|
 \*-------------------------------------*/

Raytracing::Raytracing(const Grid& grid, uint w, uint h, float dt, int nbSpheres,const MemoryType& memoryType) :
	Animable_I<uchar4>(grid, w, h, "Raytracing_Cuda_RGBA_uchar4"), memoryType(memoryType)
    {
    // Inputs
    this->dt = dt;
    this->nbSpheres = nbSpheres;

    // Tools
    this->sizeSpheres = nbSpheres * sizeof(Sphere);
    this->t = 0;

    // spheres
	{
	SphereCreator sphereCreator(nbSpheres, w, h);
	uploadToDevice(sphereCreator.getTabSphere());
	} // SphereCreator depiler, donc detruit, tabSphere cote host detruit!

    cout << "[Raytracing] : memoryType : " << memoryType;
    }

Raytracing::~Raytracing()
    {
    // TODO Raytracing GPU MemoryManagement free
    switch (memoryType)
	{
	case GM:
	    {
	    assert(false); // to delete once implement
	    // ??
	    break;
	    }
	case CM:
	    {
	    assert(false);// to delete once implement
	    // ??
	    break;
	    }
	case SM:
	    {
	    assert(false);// to delete once implement
	    // ??
	    break;
	    }
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * Override
 */
void Raytracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    // TODO Raytracing GPU lauch kernel

    switch (memoryType)
	{
	case GM:
	    {
	    assert(false);// to delete once implement
	    // Call the kernel kernelRaytacingGM (line 25)
	    break;
	    }
	case CM:
	    {
	    assert(false);// to delete once implement
	    // Call the kernel kernelRaytacingCM (line 26)
	    break;
	    }
	case SM:
	    {
	    assert(false);// to delete once implement
	    // Call the kernel kernelRaytacingSM (line 27)
	    break;
	    }
	}
    }

/**
 * Override
 */
void Raytracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

void Raytracing::uploadToDevice(Sphere* ptrTabSpheres)
    {
    // TODO Raytracing GPU MemoryManagement
    switch (memoryType)
	{
	case GM:
	    {
	    assert(false);// to delete once implement
	    // ??
	    break;
	    }
	case CM:
	    {
	    assert(false);// to delete once implement
	    // ?? use uploadToCM
	    break;
	    }
	case SM:
	    {
	    assert(false);// to delete once implement
	    // ??
	    break;
	    }
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

