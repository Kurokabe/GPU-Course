#include "hip/hip_runtime.h"
#include <assert.h>

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "Sphere.h"
#include "IndiceTools_GPU.h"

#include "RaytracingMath.h"
#include "nbSphere.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

__constant__ Sphere TAB_SPHERES_CM[NB_SPHERE]; // NB_SPHERE define in nbSphere.h

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres);
__global__ void kernelRaytacingSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres);
__global__ void kernelRaytacingCM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSpheres);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSpheres, int nbSpheres);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		host			*|
 \*-------------------------------------*/

/**
 * must be called by host
 * ptrTabSpheres est un tableau de sphere cote host
 */
__host__ void uploadToCM(Sphere* ptrTabSpheres, int nbSpheres)
    {
    assert(nbSpheres == NB_SPHERE);

    assert(false);// to delete once implement

    // TODO Raytracing GPU CM
    // mettre ptrTabSpheres dans TAB_SPHERES_CM (line 16)
    }

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres)
    {
    // TODO Raytracing GPU GM
    // call work with good input
    }

__global__ void kernelRaytacingSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres)
    {
    // TODO Raytracing GPU SM
    // call work with good input
    }

__global__ void kernelRaytacingCM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSpheres)
    {
    // TODO Raytracing GPU CM
    // call work with good input
    }

/*--------------------------------------*\
 |*		private			*|
 \*-------------------------------------*/

/**
 * Methode commune au 3 kernel ci-dessus.
 * Ici on ne sait pas si derriere ptrDevTabSpheres, c'est
 * 	- de la GM?
 * 	- de la SM?
 * 	- de la CM?
 * Pas d'importance, c'est un pointeur et on travail avec!
 */
__device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSpheres, int nbSpheres)
    {
    // TODO Raytracing GPU device side
    // create RaytracingMath
    // entrelacement
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

