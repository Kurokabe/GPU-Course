#include "hip/hip_runtime.h"
#include <assert.h>

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "Sphere.h"
#include "IndiceTools_GPU.h"

#include "RaytracingMath.h"
#include "nbSphere.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

__constant__ Sphere TAB_SPHERES_CM[NB_SPHERE]; // NB_SPHERE define in nbSphere.h

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres);
__global__ void kernelRaytacingSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres);
__global__ void kernelRaytacingCM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSpheres);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSpheres, int nbSpheres);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		host			*|
 \*-------------------------------------*/

/**
 * must be called by host
 * ptrTabSpheres est un tableau de sphere cote host
 */
__host__ void uploadToCM(Sphere* ptrTabSpheres, int nbSpheres)
    {
    assert(nbSpheres == NB_SPHERE);

    //assert(false);// to delete once implement
    Device::memcpyToCM(TAB_SPHERES_CM, ptrTabSpheres, nbSpheres * sizeof(Sphere));

    // TODO Raytracing GPU CM
    // mettre ptrTabSpheres dans TAB_SPHERES_CM (line 16)
    }

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres)
    {
    // TODO Raytracing GPU GM
    work(ptrDevPixels, w, h, t, ptrTabSpheresGM, nbSpheres);
    // call work with good input
    }

__global__ void kernelRaytacingSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres)
    {
    // TODO Raytracing GPU SM
    // call work with good input
    }

__global__ void kernelRaytacingCM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSpheres)
    {
    // TODO Raytracing GPU CM
    // call work with good input
    }

/*--------------------------------------*\
 |*		private			*|
 \*-------------------------------------*/

/**
 * Methode commune au 3 kernel ci-dessus.
 * Ici on ne sait pas si derriere ptrDevTabSpheres, c'est
 * 	- de la GM?
 * 	- de la SM?
 * 	- de la CM?
 * Pas d'importance, c'est un pointeur et on travail avec!
 */
__device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSpheres, int nbSpheres)
    {
    RaytracingMath raytracingMath = RaytracingMath(ptrDevTabSpheres, nbSpheres);
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int i;	// in [0,h[
    int j; 	// in [0,w[


    int s = TID;  // in [0,...
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); 	// update (i, j)

	raytracingMath.color(&ptrDevPixels[s], i, j, t);
	s += NB_THREAD;
	}
    // TODO Raytracing GPU device side
    // create RaytracingMath
    // entrelacement
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

