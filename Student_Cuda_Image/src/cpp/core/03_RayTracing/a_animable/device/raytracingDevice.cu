#include "hip/hip_runtime.h"
#include <assert.h>

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "Sphere.h"
#include "IndiceTools_GPU.h"

#include "RaytracingMath.h"
#include "nbSphere.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

__constant__ Sphere TAB_SPHERES_CM[NB_SPHERE]; // NB_SPHERE define in nbSphere.h

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres);
__global__ void kernelRaytacingSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres);
__global__ void kernelRaytacingCM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSpheres);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSpheres, int nbSpheres);
static __device__ void copyGMToSM(Sphere* tabGM, Sphere* tabSM, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		host			*|
 \*-------------------------------------*/

/**
 * must be called by host
 * ptrTabSpheres est un tableau de sphere cote host
 */
__host__ void uploadToCM(Sphere* ptrTabSpheres, int nbSpheres)
    {
    assert(nbSpheres == NB_SPHERE);
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(TAB_SPHERES_CM), ptrTabSpheres, nbSpheres*sizeof(Sphere), 0, hipMemcpyHostToDevice));
    }

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres)
    {
    work(ptrDevPixels, w, h, t, ptrTabSpheresGM, nbSpheres);
    }

__global__ void kernelRaytacingSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres)
    {
    extern __shared__ Sphere tabSM[];
    copyGMToSM(ptrTabSpheresGM, tabSM, nbSpheres);
    __syncthreads(); // Barrière de synchronisation des threads du block
    work(ptrDevPixels, w, h, t, tabSM, nbSpheres);
    }

__global__ void kernelRaytacingCM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSpheres)
    {
    work(ptrDevPixels, w, h, t, TAB_SPHERES_CM, nbSpheres);
    }

/*--------------------------------------*\
 |*		private			*|
 \*-------------------------------------*/

/**
 * Methode commune au 3 kernel ci-dessus.
 */
__device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSpheres, int nbSpheres)
    {
    RaytracingMath raytracingMath = RaytracingMath(ptrDevTabSpheres, nbSpheres);
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int i;	// in [0,h[
    int j; 	// in [0,w[


    int s =  Indice2D::tid();  // in [0,...
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); 	// update (i, j)

	raytracingMath.color(&ptrDevPixels[s], i, j, t);
	s += NB_THREAD;
	}
    // create RaytracingMath
    // entrelacement
    }

__device__ void copyGMToSM(Sphere* tabGM, Sphere* tabSM, int n)
    {
//    int tidLocal = Indice2D::tidLocal();
    int nbThreadLocal = Indice2D::nbThreadLocal();
    int s = Indice2D::tidLocal();
    while(s<n)
	{
	tabSM[s] = tabGM[s];
	s+=nbThreadLocal;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

