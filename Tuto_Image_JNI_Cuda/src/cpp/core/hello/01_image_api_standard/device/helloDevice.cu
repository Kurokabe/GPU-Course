#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>

#include "Indice2D.h"
#include "IndiceTools_GPU.h"
#include "cudaTools.h"
#include "Device.h"

#include "HelloMath.h"

using std::cout;
using std::endl;



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

 void launchKernelFillImageHello(uchar4* ptrDevPixels, int w, int h, int t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __global__ void kernelHello(uchar4* ptrDevPixels,int w, int h,int t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

void launchKernelFillImageHello(uchar4* ptrDevPixels, int w, int h, int t)
    {
    const int MP=Device::getMPCount();
    const int dgx=Device::getMaxThreadPerBlock();

    // Disons a optimiser
    dim3 dg(MP*4,1, 1);
    dim3 db(dgx, 1, 1);

    //Device::print(dg, db);
   //  Device::checkDimError(dg,db);

    kernelHello<<<dg,db>>>(ptrDevPixels,w,h,t);
    Device::lastCudaError("kernelHello");
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void kernelHello(uchar4* ptrDevPixels, int w, int h, int t)
    {
    HelloMath helloMath = HelloMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 colorIJ;
    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	gpu::IndiceTools::toIJ(s, w, &pixelI, &pixelJ);	 // update (pixelI, pixelJ)

	helloMath.color(&colorIJ,pixelI, pixelJ, t); 	// update color
	ptrDevPixels[s] = colorIJ;

	s += NB_THREAD;
	}

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

