#include "hip/hip_runtime.h"

#include "Bandwidth.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include "Chrono.h"


using std::cout;
using std::endl;

 /*---------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void kernelBandwidth(int* tabDataGM, int* tabFrequenceGM, int n, int dataMax);

 /*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

 /*-------------------------------------*\
 |*		Constructeur		*|
 \*-------------------------------------*/

Bandwidth::Bandwidth(const Grid& grid, int* tabData, int n) : n(n)
    {

    this->dg = grid.dg;
    this->db = grid.db;

    this->sizeTabDataGM = sizeof(int) * n; // [octet]

    Device::malloc(&tabDataGM, sizeTabDataGM);

    Chrono chrono;
    Device::memcpyHToD(tabDataGM, tabData, sizeTabDataGM);
    chrono.stop();
    elapsedTime = chrono.getElapseTimeS();
    cout << "Elapsed time : " << elapsedTime << " (s)" << endl;

    }

Bandwidth::~Bandwidth(void)
    {
    Device::free (tabDataGM);
    }

double Bandwidth::getElapsedTime()
    {
    return elapsedTime;
    }

 /*-------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Bandwidth::run()
    {
    //kernelHistogramme<<<dg,db>>>(tabDataGM);
    //Device::memcpyDToH(tabData, tabDataGM);
    }

 /*---------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
