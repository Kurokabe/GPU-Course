#include "hip/hip_runtime.h"
#include "Bandwidth.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include "Chrono.h"

using std::cout;
using std::endl;

/*---------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void kernelBandwidthEntrelacement(int* tabDataGM, int* tabDataGMCopy, int n);
extern __global__ void kernelBandwidthOneOne(int *tabDataGM,int*tabDataGMCopy, int n );

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*-------------------------------------*\
 |*		Constructeur		*|
 \*-------------------------------------*/

Bandwidth::Bandwidth(const Grid &grid, int *tabData, int n, TransferType type) :
	n(n), type(type)
    {

    this->dg = grid.dg;
    this->db = grid.db;

    this->sizeTabDataGM = sizeof(int) * n; // [octet]

    switch (type)
	{
	case TransferType::HostToDevice:
	    Device::malloc(&tabDataGM, sizeTabDataGM);
	    break;
	case TransferType::HostToDeviceDMA:
	    Device::hostMalloc(&tabDataGM, sizeTabDataGM, HostMemoryType::MAPPED_MULTIGPU);
	    break;
	case TransferType::DeviceToDevice:
	    Device::malloc(&tabDataGM, sizeTabDataGM);
	    Device::malloc(&tabDataGMCopy, sizeTabDataGM);

	    Device::memcpyHToD(tabDataGM, tabData, sizeTabDataGM);
	    break;
	case TransferType::DeviceToDeviceMultiGPUEntrelacement:
	case TransferType::DeviceToDeviceMultiGPUOneOne:
	case TransferType::DeviceToDeviceMultiGPU:
	    Device::setDevice(0);
	    Device::malloc(&tabDataGM, sizeTabDataGM);
	    Device::setDevice(1);
	    Device::malloc(&tabDataGMCopy, sizeTabDataGM);
	    Device::setDevice(0);
	    Device::memcpyHToD(tabDataGM, tabData, sizeTabDataGM);
	    break;
	case TransferType::DeviceToDeviceEntrelacement:
	case TransferType::DeviceToDeviceOneOne:
	    Device::malloc(&tabDataGM, sizeTabDataGM);
	    Device::malloc(&tabDataGMCopy, sizeTabDataGM);
	    break;
	}

    int mp = Device::getMPCount();
    int coreMP = Device::getCoreCountMP();
    dim3 dg = dim3(mp*12, 1, 1);
    dim3 db = dim3(coreMP, 1, 1);
    Chrono chrono;
    switch (type)
	{
	case TransferType::HostToDevice:
	case TransferType::HostToDeviceDMA:
	    Device::memcpyHToD(tabDataGM, tabData, sizeTabDataGM);
	    break;
	case TransferType::DeviceToDeviceMultiGPU:
	case TransferType::DeviceToDevice:
	    Device::memcpyDToD(tabDataGMCopy, tabDataGM, sizeTabDataGM);
	    Device::synchronize();
	    break;
	case TransferType::DeviceToDeviceMultiGPUEntrelacement:
	case TransferType::DeviceToDeviceEntrelacement:
	    kernelBandwidthEntrelacement<<<dg,db>>>(tabDataGM,tabDataGMCopy,n);
	    Device::synchronize();
	    break;
	case TransferType::DeviceToDeviceMultiGPUOneOne:
	case TransferType::DeviceToDeviceOneOne:
//	    dim3 dg = dim3(n/1024, 1, 1);
//	    dim3 db = dim3(1024, 1, 1);
//	    kernelBandwidthOneOne<<<dg,db>>>(tabDataGM,tabDataGMCopy,n);
	    Device::synchronize();
	    break;

	}

    chrono.stop();

    elapsedTime = chrono.getElapseTimeS();
    cout << elapsedTime<< endl;

    }

Bandwidth::~Bandwidth(void)
    {

    switch (type)
	{
	case TransferType::HostToDevice:
	    Device::free(tabDataGM);
	    break;
	case TransferType::HostToDeviceDMA:
	    Device::hostFree(tabDataGM);
	    break;
	case TransferType::DeviceToDevice:
	case TransferType::DeviceToDeviceEntrelacement:
	case TransferType::DeviceToDeviceOneOne:
	    Device::free(tabDataGM);
	    Device::free(tabDataGMCopy);
	    break;
	case TransferType::DeviceToDeviceMultiGPUEntrelacement:
	case TransferType::DeviceToDeviceMultiGPUOneOne:
	case TransferType::DeviceToDeviceMultiGPU:
	    Device::free(tabDataGM);
	    Device::setDevice(1);
	    Device::free(tabDataGMCopy);
	    Device::setDevice(0);
	    break;
	}
    }

double Bandwidth::getElapsedTime()
    {
    return elapsedTime;
    }

/*-------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Bandwidth::run()
    {
    //kernelHistogramme<<<dg,db>>>(tabDataGM);
    //Device::memcpyDToH(tabData, tabDataGM);
    }

/*---------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
