#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void kernelBandwidthEntrelacement(int *tabDataGM,int*tabDataGMCopy, int n );
__global__ void kernelBandwidthOneOne(int *tabDataGM,int*tabDataGMCopy, int n );

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void kernelBandwidthEntrelacement(int *tabDataGM,int*tabDataGMCopy, int n )
    {
    const int NB_THREAD = Indice2D::nbThread();
        const int TID = Indice2D::tid();

        int s = TID;
        while (s < n)
    	{
    	tabDataGMCopy[s]=tabDataGM[s];
    	s += NB_THREAD;
    	}
    }

__global__ void kernelBandwidthOneOne(int *tabDataGM,int*tabDataGMCopy, int n )
    {
        const int TID = Indice2D::tid();


    	tabDataGMCopy[TID]=tabDataGM[TID];


    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
