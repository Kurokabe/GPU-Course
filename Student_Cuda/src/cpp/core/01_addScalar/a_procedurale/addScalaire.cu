#include "hip/hip_runtime.h"
// Attention : Extension .cu

#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ int addScalarGPU(int a, int b); 			// __host__ facultatif

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ static void addScalar(int a, int b, int* ptrDevC);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ int addScalarGPU(int a, int b) // __host__ facultatif
    {
    int c;
    int* ptrC = &c;	// on host (CPU)
    int* ptrDevC; 	// on device (GPU)

    // Specifier nb thread : ici 1 thread au total !
    dim3 dg = dim3(1, 1, 1);
    dim3 db = dim3(1, 1, 1);

    //Device::gridHeuristic(dg, db);

    size_t size = sizeof(int); // [octet]
    Device::malloc(&ptrDevC, size); // Device memory allocation (*)

    addScalar<<<dg,db>>>(a,b,ptrDevC); // asynchrone !!

    //v1
    Device::synchronize(); // Debug : necessaire only pour printf sur device

    //v2
    // hipDeviceSynchronize(); // Debug : necessaire only pour printf sur device

    // memoryManagement => barrier de synchronisation
    Device::memcpyDToH(ptrC, ptrDevC, size);   // Device -> Host
    Device::free(ptrDevC); //  dispose memory create in (*)

    return c;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * Hyp : 1 seul thread, ie dg(1,1,1) et db (1,1,1)
 */
__global__ void addScalar(int a, int b, int* ptrDevC)
    {
    *ptrDevC = a + b; // Hyp: 1 seul thread

    // debug
    printf("[GPU] %d + %d = %d", a, b, *ptrDevC);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

