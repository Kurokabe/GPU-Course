#include "hip/hip_runtime.h"
#include "AddScalar.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addScalar(float a, float b, float* ptrDevC);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddScalar::AddScalar(float a, float b, float* ptrC) :
	a(a), b(b), ptrC(ptrC)
    {
    this->sizeFloat = sizeof(float); // [octet]

    // MM
	{
	Device::malloc(&ptrDevC, sizeFloat);
	}
    }

AddScalar::~AddScalar(void)
    {
    //MM
	{
	Device::free (ptrDevC);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void AddScalar::run()
    {
    // 1 seul thread suffit
    dim3 dg(1,1,1);
    dim3 db(1,1,1);
    assert(dg.x*dg.y*dg.z*db.x*db.y*db.z==1);

    addScalar<<<dg,db>>>(a, b, ptrDevC);// assynchrone

    //Device::synchronize(); // Debug : necessaire only pour printf sur device

    // MM (Device -> Host)
	{
	Device::memcpyDToH(ptrC, ptrDevC, sizeFloat); // barriere synchronisation implicite
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
