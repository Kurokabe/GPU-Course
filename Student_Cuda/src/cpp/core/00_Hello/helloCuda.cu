#include "hip/hip_runtime.h"
// Attention : Extension .cu

#include <iostream>
#include "cudaTools.h"
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ bool helloCuda(void); //__host__ facultatif

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ static void kernelHello(void);
__device__ static void doSomethingHello(void);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * http://www.icl.utk.edu/~mgates3/docs/cuda.html
 */
__host__ bool helloCuda(void) //__host__ facultatif
    {
    cout << endl << "[Hello Cuda] : kernel empty" << endl;

    // Specifier nb thread : ici 1 thread au total !
    dim3 dg = dim3(1, 1, 1);
    dim3 db = dim3(1, 1, 1);

    //Device::gridHeuristic(dg, db);

    kernelHello<<<dg,db>>>();  // asynchrone !!

    return true;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * output : void
 */
__global__ void kernelHello(void)
    {
    doSomethingHello();
    }

/**
 * Can be call only by device
 * inliner by nvcc (nvidia compiler)
 */
__device__ void doSomethingHello(void)
    {
    // rien
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

