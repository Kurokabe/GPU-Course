#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void reductionIntraThread(float* tabGM, int nbSlice);
__global__ void ecrasementGM(float* tabGM, int moitier);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ float f(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * Chaque thread effecteur une reduction avec le patern d'entrelacement,
 * puis stocke son résultat dans SA case dans tabGM
 * tabGM est un tableau promu, qui a autant de case que de thread
 */
__global__ void reductionIntraThread(float* tabGM, int nbSlice)
    {
    const int TID=threadIdx.x+(blockIdx.x*blockDim.x) ; //global à la grille
    const int NB_THREAD= blockDim.x*gridDim.x; //nbThreadTotal

    const float DX = 1/(float)nbSlice;
    int s=TID;
    float sum = 0;
    while(s<nbSlice)
	{
	float xs = s*DX;
	sum+=f(xs);
	s+=NB_THREAD;
	}
    tabGM[TID]=sum;
    }

/**
 * Pour version NaivePLus
 *
 * Ramene la semantique du tableau pour une reduction, dans la premiere moitier, par ecraseemt
 * Doit etre appeler plusieurs fois coter host, pour tout ramener dans tabGM[0]
 */
__global__ void ecrasementGM(float* tabGM, int moitier)
    {
    const int TID=threadIdx.x+(blockIdx.x*blockDim.x);
    tabGM[TID] += tabGM[TID + moitier];
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float f(float x)
    {
    return 4/(1+x*x);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

