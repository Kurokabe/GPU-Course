#include "hip/hip_runtime.h"
#include "../../04_Slice/host/Slice.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void reductionIntraThread(float* tabGM,int nbSlice);
extern __global__ void ecrasementGM(float* tabGM, int moitier);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Slice::Slice(const Grid& grid, int nbSlice, float* ptrPiHat) :
	ptrPiHat(ptrPiHat), nbSlice(nbSlice)
    {
    this->nTabGM = grid.threadCounts();
    this->sizeTabGM = nTabGM * sizeof(float); // [octet]

    // MM
	{
	//MM(malloc Device)
	    {
	    Device::malloc(&tabGM, sizeTabGM);
	    }
	}

    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}
    }

Slice::~Slice(void)
    {
    //MM (device free)
	{
	Device::free(tabGM);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Slice::run()
    {
    // v1 : promotion tab, reduction coter host
	{
	// emploi uniquemnet du kernel : reductionIntraThread

	reductionIntraThread<<<dg, db>>>(tabGM, nbSlice);

	// Reduction paralle sur cpu du tableau promu ramener coter host
	    {

	    int moitier = nTabGM/2;
	    dg.x = moitier;
	    db.x = 1;
	    while(moitier>=1)
		{
		ecrasementGM<<<dg, db>>>(tabGM, moitier);
		moitier/=2;
		dg.x=moitier;
		}
	    }
	    //MM (Device -> Host)
	    {
	    Device::memcpyDToH(ptrPiHat, tabGM,sizeof(float));
	    }
	}

    // v2 : promotion tab, reduction coter device en GM
	{
	// emploi de deux kernels:
	//	reductionIntraThread
	//	ecrasementGM
	//
	// Warning:
	//	(W1) ecrasementGM doit etre lancer dans une boucle coter host
	//	(W2) On serait tenter de realiser cette boucle coter device, c'est techniquement possible, mais pose de gros problème de synchronisation!
	//
	// Note:
	//	Attendez la version de slice en SM pour un code 100% parallel sur GPU, sans boucle sur CPU.

	// TODO Slice+
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
