#include "hip/hip_runtime.h"
#include "../../04_Slice/host/Slice.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void reductionIntraThread(float* tabGM,int nbSlice);
extern __global__ void ecrasementGM(float* tabGM, int moitier);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Slice::Slice(const Grid& grid, int nbSlice, float* ptrPiHat) :
	ptrPiHat(ptrPiHat), nbSlice(nbSlice)
    {
    this->nTabGM = grid.threadCounts();
    this->sizeTabGM = nTabGM * sizeof(float); // [octet]

    // MM
	{
	//MM(malloc Device)
	    {
	    Device::malloc(&tabGM, sizeTabGM);
	    }
	}

    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}
    }

Slice::~Slice(void)
    {
    //MM (device free)
	{
	Device::free(tabGM);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Slice::run()
    {

    reductionIntraThread<<<dg, db>>>(tabGM, nbSlice);

    // Reduction paralle sur cpu du tableau promu ramener coter host
	{
	int moitier = nTabGM/2;
	dg.x = moitier;
	db.x = 1;
	while(moitier>=1)
	    {
	    ecrasementGM<<<dg, db>>>(tabGM, moitier);
	    moitier/=2;
	    dg.x=moitier;
	    }
	}
	//MM (Device -> Host)
	{
	Device::memcpyDToH(ptrPiHat, tabGM,sizeof(float));
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
