#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void addVector(float* ptrDevV1, float* ptrDevV2, float* ptrDevW, int n);
__global__ void addVector11(float* ptrDevV1, float* ptrDevV2, float* ptrDevW, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 * pattern entrelacement
 */
__global__ void addVector(float* ptrDevV1, float* ptrDevV2, float* ptrDevW, int n)
    {
    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();

    // Debug, facultatif
//    if (TID == 0)
//	{
//	printf("Coucou from device tid = %d", TID); //required   Device::synchronize(); after the call of kernel
//	}

    //TODO addVector pattern entrelacement
    }

/**
 * pattern 1<-->1
 * hyp: #thread=#caseVecteur=n
 */
__global__ void addVector11(float* ptrDevV1, float* ptrDevV2, float* ptrDevW, int n)
    {
    // TODO addVector pattern  1<-->1
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

