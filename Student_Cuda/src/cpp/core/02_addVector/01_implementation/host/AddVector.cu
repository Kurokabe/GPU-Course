#include "hip/hip_runtime.h"
#include "AddVector.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addVector(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);
extern __global__ void addVector11(float* ptrDevV1, float* ptrDevV2, float* ptrDevW, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddVector::AddVector(const Grid& grid, float* ptrV1, float* ptrV2, float* ptrW, int n) :
	ptrV1(ptrV1), ptrV2(ptrV2), ptrW(ptrW), n(n)
    {
    this->sizeVector= n * sizeof(float); // octet

    // MM
	{

	// MM (malloc Device)
	    {
	    Device::malloc(&ptrDevV1, sizeVector);
	    Device::malloc(&ptrDevV2, sizeVector);
	    Device::malloc(&ptrDevW, sizeVector);
	    }

	// MM (copy Host->Device)
	    {
	    Device::memcpyHToD(ptrDevV1, ptrV1, sizeVector);
	    Device::memcpyHToD(ptrDevV2, ptrV2, sizeVector);
	    }

	}

    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}
    }

AddVector::~AddVector(void)
    {
    //MM (device free)
	{
	Device::free(ptrDevV1);
	Device::free(ptrDevV2);
	Device::free(ptrDevW);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void AddVector::run()
    {
    //v1 pattern entrelacement
    addVector<<<dg,db>>>(ptrDevV1, ptrDevV2, ptrDevW, n);// assynchrone

    //v2 pattern 1<-->1
	{
	//assert(db.x*db.y*db.z*dg.x*dg.y*dg.z==n);
	//addVector11<<<dg,db>>>(ptrDevV1, ptrDevV2, ptrDevW, n); // assynchrone
	}

    //Device::synchronize(); // Temp,debug, only for printf in  GPU, sinon pas necessaire

    // MM (Device -> Host)
	{
	Device::memcpyDToH(ptrW, ptrDevW, sizeVector);
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
