#include "hip/hip_runtime.h"

#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "ReductionAddTools.h"
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void kernelSliceSM(float* ptrDevResultat, int nbSlice);
__device__ void reductionIntraThreadSM(float* tabSM, int nbSlice);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ float f(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void kernelSliceSM(float* ptrDevResultat, int nbSlice)
    {
    extern __shared__ float tabSM[];
    reductionIntraThreadSM(tabSM, nbSlice);
    __syncthreads();
    ReductionAddTools::reductionADD(tabSM, ptrDevResultat);
    }
/**
 * Chaque thread effecteur une reduction avec le patern d'entrelacement,
 * puis stocke son résultat dans SA case dans tabGM
 * tabGM est un tableau promu, qui a autant de case que de thread
 */

__device__ void reductionIntraThreadSM(float* tabSM, int nbSlice)
    {
    const int TID = threadIdx.x + (blockIdx.x * blockDim.x); //global à la grille
    const int TID_LOCAL = threadIdx.x;
    const int NB_THREAD = blockDim.x * gridDim.x; //nbThreadTotal

    const float dx = 1 / (float) nbSlice;
    int s = TID;
    float sum = 0;
    while (s < nbSlice)
	{
	float xs = s * dx;
	sum += f(xs);
	s += NB_THREAD;
	}
    tabSM[TID_LOCAL] = sum;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float f(float x)
    {
    return 4 / (1 + x * x);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
