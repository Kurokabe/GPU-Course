#include "hip/hip_runtime.h"
#include <iostream>

#include "ReductionAddToolsLock.h"
#include "Grid.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration					*|
 \*---------------------------------------------------------------------*/

/*---------------------*\
|*	public		*|
 \*----------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddToolsLock_II_Ok(const Grid& grid); // __host__ facultatif

/*---------------------*\
|*	private		*|
 \*----------------------*/

static __global__ void fillTidGlobal(long* ptrDevResultGM);
static __device__ void reductionIntraThread(long* tabSM);

/*----------------------------------------------------------------------*\
|*			Host	 					*|
 \*---------------------------------------------------------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddToolsLock_II_Ok(const Grid& grid)
    {
    // TODO
    // MM pour ptrDevResultGM (oubliez pas initialisation)
    // appeler kernel
    // MM recuprer resultat
    // cheker resultat
    }

/*----------------------------------------------------------------------*\
|*			Device	 					*|
 \*---------------------------------------------------------------------*/

/**
 * protocole :	TID_GLOBAL partout en tabSM
 * resultat:	 (n-1)n/2 ou n=NB_THREAD_GLOBAL
 */
__global__ void fillTidGlobal(long* ptrDevResultGM)
    {
    // TODO declaration tabSM
    // reductionIntraThread
    // ReductionAddToolsLock

    // __syncthreads(); // des threads de meme block!// utile? ou?
    }

__device__ void reductionIntraThread(long* tabSM)
    {
    // TODO entrelacement et remplissage tabSM
    // rappel : |tabSM|=|threadByBlock|
    }

/*----------------------------------------------------------------------*\
|*			End	 					*|
 \*---------------------------------------------------------------------*/
