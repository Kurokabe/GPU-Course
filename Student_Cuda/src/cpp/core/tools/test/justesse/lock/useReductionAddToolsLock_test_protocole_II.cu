#include "hip/hip_runtime.h"
#include <iostream>

#include "ReductionAddToolsLock.h"
#include "Grid.h"

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration					*|
 \*---------------------------------------------------------------------*/

/*---------------------*\
|*	public		*|
 \*----------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddToolsLock_II_Ok(const Grid& grid); // __host__ facultatif
__host__ bool isReductionAddToolsLock_II_Ok(); // __host__ facultatif

/*---------------------*\
|*	private		*|
 \*----------------------*/

static __global__ void fillTidGlobal(long* ptrDevResultGM);
static __device__ void reductionIntraThread(long* tabSM);

/*----------------------------------------------------------------------*\
|*			Host	 					*|
 \*---------------------------------------------------------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddToolsLock_II_Ok(const Grid& grid)
    {
    // MM pour ptrDevResultGM (oubliez pas initialisation)
    // appeler kernel
    // MM recuprer resultat
    // cheker resultat
    long* ptrRes = new long[sizeof(long)];
    long* ptrResGM;
    *ptrRes = 0;
    Device::malloc(&ptrResGM, sizeof(long));
    Device::memcpyHToD(ptrResGM, ptrRes, sizeof(long));
    fillTidGlobal<<<grid.dg, grid.db,sizeof(long)*grid.db.x>>>(ptrResGM);
    Device::memcpyDToH(ptrRes, ptrResGM, sizeof(long));
    long n = grid.db.x * grid.dg.x;
    return * ptrRes == n/2*(n-1);
    }


__host__ bool isReductionAddToolsLock_II_Ok()
    {
    bool isOk = true;
    dim3 dg = dim3(1,1,1);
    dim3 db = dim3(2, 1, 1);
    Grid grid(dg, db);
    for(int i = 1; i<=64; ++i)
    	{
    	dg.x = i;
    	grid.dg = dg;
    	for(int j = 2; j<=1024; j*=2)
    	    {
    	    db.x = j;
    	    grid.db = db;
    	    cout << grid << endl;
    	    isOk &= isReductionAddToolsLock_II_Ok(grid);
    	    }
    	}
    return isOk;
    }
/*----------------------------------------------------------------------*\
|*			Device	 					*|
 \*---------------------------------------------------------------------*/

/**
 * protocole :	TID_GLOBAL partout en tabSM
 * resultat:	 (n-1)n/2 ou n=NB_THREAD_GLOBAL
 */
__global__ void fillTidGlobal(long* ptrDevResultGM)
    {
    // TODO declaration tabSM
    // reductionIntraThread
    // ReductionAddToolsLock
    extern __shared__ long tabSM[];
    reductionIntraThread(tabSM);
    __syncthreads();
    ReductionAddToolsLock::reductionADD(tabSM, ptrDevResultGM);
    // __syncthreads(); // des threads de meme block!// utile? ou?
    }

__device__ void reductionIntraThread(long* tabSM)
    {
    // TODO entrelacement et remplissage tabSM
    // rappel : |tabSM|=|threadByBlock|
    tabSM[threadIdx.x] = blockDim.x * blockIdx.x + threadIdx.x;
    }

/*----------------------------------------------------------------------*\
|*			End	 					*|
 \*---------------------------------------------------------------------*/
