#include "hip/hip_runtime.h"
#include <iostream>

#include "ReductionAddToolsLock.h"
#include "Grid.h"

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration					*|
 \*---------------------------------------------------------------------*/

/*---------------------*\
|*	public		*|
 \*----------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddToolsLock_II_Ok(const Grid& grid); // __host__ facultatif

/*---------------------*\
|*	private		*|
 \*----------------------*/

static __global__ void fillTidGlobal(long* ptrDevResultGM);
static __device__ void reductionIntraThread(long* tabSM);

/*----------------------------------------------------------------------*\
|*			Host	 					*|
 \*---------------------------------------------------------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddToolsLock_II_Ok(const Grid& grid)
    {
    // TODO
    // MM pour ptrDevResultGM (oubliez pas initialisation)
    // appeler kernel
    // MM recuprer resultat
    // cheker resultat
    long* ptrRes = new long[sizeof(long)];
    long* ptrResGM;
    *ptrRes = 0;
    Device::malloc(&ptrResGM, sizeof(long));
    Device::memcpyHToD(ptrResGM, ptrRes, sizeof(long));
    fillTidGlobal<<<grid.dg, grid.db,sizeof(long)*grid.db.x>>>(ptrResGM);
    Device::memcpyDToH(ptrRes, ptrResGM, sizeof(long));
    long n = grid.db.x * grid.dg.x;
    return * ptrRes == n/2*(n-1);
    }

/*----------------------------------------------------------------------*\
|*			Device	 					*|
 \*---------------------------------------------------------------------*/

/**
 * protocole :	TID_GLOBAL partout en tabSM
 * resultat:	 (n-1)n/2 ou n=NB_THREAD_GLOBAL
 */
__global__ void fillTidGlobal(long* ptrDevResultGM)
    {
    // TODO declaration tabSM
    // reductionIntraThread
    // ReductionAddToolsLock
    extern __shared__ long tabSM[];
    reductionIntraThread(tabSM);
    __syncthreads();
    ReductionAddToolsLock::reductionADD(tabSM, ptrDevResultGM);
    // __syncthreads(); // des threads de meme block!// utile? ou?
    }

__device__ void reductionIntraThread(long* tabSM)
    {
    // TODO entrelacement et remplissage tabSM
    // rappel : |tabSM|=|threadByBlock|
    tabSM[threadIdx.x] = blockDim.x * blockIdx.x + threadIdx.x;
    }

/*----------------------------------------------------------------------*\
|*			End	 					*|
 \*---------------------------------------------------------------------*/
