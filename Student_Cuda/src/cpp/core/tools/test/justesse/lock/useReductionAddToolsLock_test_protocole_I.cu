#include "hip/hip_runtime.h"
#include <iostream>

#include "ReductionAddToolsLock.h"
#include "Grid.h"

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration					*|
 \*---------------------------------------------------------------------*/

/*---------------------*\
|*	public		*|
 \*----------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddToolsLock_I_Ok(const Grid& grid); // __host__ facultatif
__host__ bool isReductionAddToolsLock_I_Ok(); // __host__ facultatif

/*---------------------*\
|*	private		*|
 \*----------------------*/

static __global__ void fillOne(long* ptrDevResultGM);
static __device__ void reductionIntraThread(long* tabSM);

/*----------------------------------------------------------------------*\
|*			Host	 					*|
 \*---------------------------------------------------------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddToolsLock_I_Ok(const Grid& grid)
    {
    // TODO
    // MM pour ptrDevResultGM (oubliez pas initialisation)
    // appeler kernel
    // MM recuprer resultat
    // cheker resultat
    long* ptrRes = new long[sizeof(long)];
    long* ptrResGM;
    *ptrRes = 0;
    Device::malloc(&ptrResGM, sizeof(long));
    Device::memcpyHToD(ptrResGM, ptrRes, sizeof(long));
    fillOne<<<grid.dg, grid.db,sizeof(long)*grid.db.x>>>(ptrResGM);
    Device::memcpyDToH(ptrRes, ptrResGM, sizeof(long));
    return * ptrRes == grid.db.x * grid.dg.x;
    }

__host__ bool isReductionAddToolsLock_I_Ok()
    {
    bool isOk = true;
    dim3 dg = dim3(1,1,1);
    dim3 db = dim3(2, 1, 1);
    Grid grid(dg, db);
    for(int i = 1; i<=64; ++i)
	{
	dg.x = i;
	grid.dg = dg;
	for(int j = 2; j<=1024; j*=2)
	    {
	    db.x = j;
	    grid.db = db;
	    cout << grid << endl;
	    isOk &= isReductionAddToolsLock_I_Ok(grid);
	    }
	}
    return isOk;
    }

/*----------------------------------------------------------------------*\
|*			Device	 					*|
 \*---------------------------------------------------------------------*/

/**
 * protocole :	1 partout en tabSM
 * resultat:	NB_THREAD_GLOBAL
 */
__global__ void fillOne(long* ptrDevResultGM)
    {
    // TODO declaration tabSM
    // reductionIntraThread
    // ReductionAddToolsLock
    extern __shared__ long tabSM[];
    reductionIntraThread(tabSM);
    __syncthreads();
    ReductionAddToolsLock::reductionADD(tabSM, ptrDevResultGM);
    // __syncthreads(); // des threads de meme block!// utile? ou?
    }

__device__ void reductionIntraThread(long* tabSM)
    {
    // TODO entrelacement et remplissage tabSM
    // rappel : |tabSM|=|threadByBlock|
    tabSM[threadIdx.x] = 1;
    }

/*----------------------------------------------------------------------*\
|*			End	 					*|
 \*---------------------------------------------------------------------*/
