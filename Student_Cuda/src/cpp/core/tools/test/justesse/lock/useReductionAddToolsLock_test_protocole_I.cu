#include "hip/hip_runtime.h"
#include <iostream>

#include "ReductionAddToolsLock.h"
#include "Grid.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration					*|
 \*---------------------------------------------------------------------*/

/*---------------------*\
|*	public		*|
 \*----------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddToolsLock_I_Ok(const Grid& grid); // __host__ facultatif

/*---------------------*\
|*	private		*|
 \*----------------------*/

static __global__ void fillOne(long* ptrDevResultGM);
static __device__ void reductionIntraThread(long* tabSM);

/*----------------------------------------------------------------------*\
|*			Host	 					*|
 \*---------------------------------------------------------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddToolsLock_I_Ok(const Grid& grid)
    {
    // TODO
    // MM pour ptrDevResultGM (oubliez pas initialisation)
    // appeler kernel
    // MM recuprer resultat
    // cheker resultat
    }

/*----------------------------------------------------------------------*\
|*			Device	 					*|
 \*---------------------------------------------------------------------*/

/**
 * protocole :	1 partout en tabSM
 * resultat:	NB_THREAD_GLOBAL
 */
__global__ void fillOne(long* ptrDevResultGM)
    {
    // TODO declaration tabSM
    // reductionIntraThread
    // ReductionAddToolsLock

    // __syncthreads(); // des threads de meme block!// utile? ou?
    }

__device__ void reductionIntraThread(long* tabSM)
    {
    // TODO entrelacement et remplissage tabSM
    // rappel : |tabSM|=|threadByBlock|
    }

/*----------------------------------------------------------------------*\
|*			End	 					*|
 \*---------------------------------------------------------------------*/
