#include "hip/hip_runtime.h"
#include <iostream>

#include "ReductionAddToolsLock.h"
#include "Grid.h"

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration					*|
 \*---------------------------------------------------------------------*/

/*---------------------*\
|*	public		*|
 \*----------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddToolsLock_I_Ok(const Grid& grid); // __host__ facultatif

/*---------------------*\
|*	private		*|
 \*----------------------*/

static __global__ void fillOne(long* ptrDevResultGM);
static __device__ void reductionIntraThread(long* tabSM);

/*----------------------------------------------------------------------*\
|*			Host	 					*|
 \*---------------------------------------------------------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddToolsLock_I_Ok(const Grid& grid)
    {
    // TODO
    // MM pour ptrDevResultGM (oubliez pas initialisation)
    // appeler kernel
    // MM recuprer resultat
    // cheker resultat
    long* ptrRes = new long[sizeof(long)];
    long* ptrResGM;
    *ptrRes = 0;
    Device::malloc(&ptrResGM, sizeof(long));
    Device::memcpyHToD(ptrResGM, ptrRes, sizeof(long));
    fillOne<<<grid.dg, grid.db,sizeof(long)*grid.db.x>>>(ptrResGM);
    Device::memcpyDToH(ptrRes, ptrResGM, sizeof(long));
    return * ptrRes == grid.db.x * grid.dg.x;
    }

/*----------------------------------------------------------------------*\
|*			Device	 					*|
 \*---------------------------------------------------------------------*/

/**
 * protocole :	1 partout en tabSM
 * resultat:	NB_THREAD_GLOBAL
 */
__global__ void fillOne(long* ptrDevResultGM)
    {
    // TODO declaration tabSM
    // reductionIntraThread
    // ReductionAddToolsLock
    extern __shared__ long tabSM[];
    reductionIntraThread(tabSM);
    __syncthreads();
    ReductionAddToolsLock::reductionADD(tabSM, ptrDevResultGM);
    // __syncthreads(); // des threads de meme block!// utile? ou?
    }

__device__ void reductionIntraThread(long* tabSM)
    {
    // TODO entrelacement et remplissage tabSM
    // rappel : |tabSM|=|threadByBlock|
    tabSM[threadIdx.x] = 1;
    }

/*----------------------------------------------------------------------*\
|*			End	 					*|
 \*---------------------------------------------------------------------*/
