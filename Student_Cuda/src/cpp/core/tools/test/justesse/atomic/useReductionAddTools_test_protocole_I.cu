#include "hip/hip_runtime.h"
#include <iostream>

#include "ReductionAddTools.h"
#include "Grid.h"

#include "Device.h"


using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration					*|
 \*---------------------------------------------------------------------*/

/*---------------------*\
|*	public		*|
 \*----------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddTools_I_Ok(const Grid& grid); // __host__ facultatif

/*---------------------*\
|*	private		*|
 \*----------------------*/

static __global__ void fillOne(int* ptrDevResultGM);
static __device__ void reductionIntraThread(int* tabSM);

/*----------------------------------------------------------------------*\
|*			Host	 					*|
 \*---------------------------------------------------------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddTools_I_Ok(const Grid& grid)
    {
    // TODO
    // MM pour ptrDevResultGM (oubliez pas initialisation)
    // appeler kernel
    // MM recuprer resultat
    // cheker resultat
    int* ptrRes = new int[sizeof(int)];
    int* ptrResGM;
    *ptrRes = 0;
    Device::malloc(&ptrResGM, sizeof(int));

    Device::memcpyHToD(ptrResGM, ptrRes, sizeof(int));
    fillOne<<<grid.dg, grid.db, sizeof(int)*grid.db.x>>>(ptrResGM);
    Device::memcpyDToH(ptrRes, ptrResGM, sizeof(int));
    return *ptrRes == grid.db.x * grid.dg.x;
    }

/*----------------------------------------------------------------------*\
|*			Device	 					*|
 \*---------------------------------------------------------------------*/

/**
 * protocole :	1 partout en tabSM
 * resultat:	NB_THREAD_GLOBAL
 */
__global__ void fillOne(int* ptrDevResultGM)
    {
    // reductionIntraThread
    // ReductionAddTools

    extern __shared__ int tabSM[];
    reductionIntraThread(tabSM);
    __syncthreads();
    ReductionAddTools::reductionADD(tabSM, ptrDevResultGM);
    }

__device__ void reductionIntraThread(int* tabSM)
    {
    // rappel : |tabSM|=|threadByBlock|const int tid = blockDim.x * blockIdx threadIdx.x
    tabSM[threadIdx.x] = 1;
    }

/*----------------------------------------------------------------------*\
|*			End	 					*|
 \*---------------------------------------------------------------------*/
