#include "hip/hip_runtime.h"
#include <iostream>

#include "ReductionAddTools.h"
#include "Grid.h"

#include "Device.h"


using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration					*|
 \*---------------------------------------------------------------------*/

/*---------------------*\
|*	public		*|
 \*----------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddTools_I_Ok(const Grid& grid); // __host__ facultatif

/*---------------------*\
|*	private		*|
 \*----------------------*/

static __global__ void fillOne(int* ptrDevResultGM);
static __device__ void reductionIntraThread(int* tabSM);

/*----------------------------------------------------------------------*\
|*			Host	 					*|
 \*---------------------------------------------------------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddTools_I_Ok(const Grid& grid)
    {
    // TODO
    // MM pour ptrDevResultGM (oubliez pas initialisation)
    // appeler kernel
    // MM recuprer resultat
    // cheker resultat
    int* ptrRes = new int[sizeof(int)];
    int* ptrResGM;
    *ptrRes = 0;
    Device::malloc(&ptrResGM, sizeof(int));

    Device::memcpyHToD(ptrResGM, ptrRes, sizeof(int));
    fillOne<<<grid.dg, grid.db, sizeof(int)*grid.db.x>>>(ptrResGM);
    Device::memcpyDToH(ptrRes, ptrResGM, sizeof(int));
    return *ptrRes == grid.db.x * grid.dg.x;
    }

__host__ bool isReductionAddTools_I_Ok()
    {
    bool isOk = true;
    dim3 dg = dim3(1,1,1);
    dim3 db = dim3(2, 1, 1);
    Grid grid(dg, db);
    for(int i = 1; i<=64; ++i)
	{
	dg.x = i;
	grid.dg = dg;
	for(int j = 2; j<=1024; j*=2)
	    {
	    db.x = j;
	    grid.db = db;
	    cout << grid << endl;
	    isOk &= isReductionAddTools_I_Ok(grid);
	    }
	}
    }

/*----------------------------------------------------------------------*\
|*			Device	 					*|
 \*---------------------------------------------------------------------*/

/**
 * protocole :	1 partout en tabSM
 * resultat:	NB_THREAD_GLOBAL
 */
__global__ void fillOne(int* ptrDevResultGM)
    {
    // reductionIntraThread
    // ReductionAddTools

    extern __shared__ int tabSM[];
    reductionIntraThread(tabSM);
    __syncthreads();
    ReductionAddTools::reductionADD(tabSM, ptrDevResultGM);
    }

__device__ void reductionIntraThread(int* tabSM)
    {
    // rappel : |tabSM|=|threadByBlock|const int tid = blockDim.x * blockIdx threadIdx.x
    tabSM[threadIdx.x] = 1;
    }

/*----------------------------------------------------------------------*\
|*			End	 					*|
 \*---------------------------------------------------------------------*/
