#include "hip/hip_runtime.h"
#include <iostream>

#include "ReductionAddTools.h"
#include "Grid.h"


#include "Device.h"


using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration					*|
 \*---------------------------------------------------------------------*/

/*---------------------*\
|*	public		*|
 \*----------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddTools_II_Ok(const Grid& grid); // __host__ facultatif

/*---------------------*\
|*	private		*|
 \*----------------------*/

static __global__ void fillTidGlobal(int* ptrDevResultGM);
static __device__ void reductionIntraThread(int* tabSM);

/*----------------------------------------------------------------------*\
|*			Host	 					*|
 \*---------------------------------------------------------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddTools_II_Ok(const Grid& grid)
    {
    // MM pour ptrDevResultGM (oubliez pas initialisation)
    // appeler kernel
    // MM recuprer resultat
    // cheker resultat
    int* ptrRes = new int[sizeof(int)];
    int* ptrResGM;
    *ptrRes = 0;
    Device::malloc(&ptrResGM, sizeof(int));

    Device::memcpyHToD(ptrResGM, ptrRes, sizeof(int));
    fillTidGlobal<<<grid.dg, grid.db, sizeof(int)*grid.db.x>>>(ptrResGM);
    Device::memcpyDToH(ptrRes, ptrResGM, sizeof(int));
    long n = grid.db.x * grid.dg.x;
    return *ptrRes == n/2*(n-1);
    }

/*----------------------------------------------------------------------*\
|*			Device	 					*|
 \*---------------------------------------------------------------------*/

/**
 * protocole :	TID_GLOBAL partout en tabSM
 * resultat:	 (n-1)n/2 ou n=NB_THREAD_GLOBAL
 */
__global__ void fillTidGlobal(int* ptrDevResultGM)
    {
    // TODO declaration tabSM
    // reductionIntraThread
    // ReductionAddTools
    extern __shared__ int tabSM[];
    reductionIntraThread(tabSM);
    __syncthreads();
    ReductionAddTools::reductionADD(tabSM, ptrDevResultGM);

    // __syncthreads(); // des threads de meme block!// utile? ou?
    }

__device__ void reductionIntraThread(int* tabSM)
    {
    // TODO entrelacement et remplissage tabSM
    // rappel : |tabSM|=|threadByBlock|
    tabSM[threadIdx.x] = blockDim.x * blockIdx.x + threadIdx.x;
    }

/*----------------------------------------------------------------------*\
|*			End	 					*|
 \*---------------------------------------------------------------------*/
