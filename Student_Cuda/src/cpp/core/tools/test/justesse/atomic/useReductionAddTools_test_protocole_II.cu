#include "hip/hip_runtime.h"
#include <iostream>

#include "ReductionAddTools.h"
#include "Grid.h"


#include "Device.h"


using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration					*|
 \*---------------------------------------------------------------------*/

/*---------------------*\
|*	public		*|
 \*----------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddTools_II_Ok(const Grid& grid); // __host__ facultatif
__host__ bool isReductionAddTools_II_Ok(); // __host__ facultatif

/*---------------------*\
|*	private		*|
 \*----------------------*/

static __global__ void fillTidGlobal(int* ptrDevResultGM);
static __device__ void reductionIntraThread(int* tabSM);

/*----------------------------------------------------------------------*\
|*			Host	 					*|
 \*---------------------------------------------------------------------*/

/**
 * contrainte : db puissance de 2
 */
__host__ bool isReductionAddTools_II_Ok(const Grid& grid)
    {
    // MM pour ptrDevResultGM (oubliez pas initialisation)
    // appeler kernel
    // MM recuprer resultat
    // cheker resultat
    int* ptrRes = new int[sizeof(int)];
    int* ptrResGM;
    *ptrRes = 0;
    Device::malloc(&ptrResGM, sizeof(int));

    Device::memcpyHToD(ptrResGM, ptrRes, sizeof(int));
    fillTidGlobal<<<grid.dg, grid.db, sizeof(int)*grid.db.x>>>(ptrResGM);
    Device::memcpyDToH(ptrRes, ptrResGM, sizeof(int));
    long n = grid.db.x * grid.dg.x;
    return *ptrRes == n/2*(n-1);
    }

__host__ bool isReductionAddTools_II_Ok()
    {
    bool isOk = true;
    dim3 dg = dim3(1,1,1);
    dim3 db = dim3(2, 1, 1);
    Grid grid(dg, db);
    for(int i = 1; i<=64; ++i)
    	{
    	dg.x = i;
    	grid.dg = dg;
    	for(int j = 2; j<=1024; j*=2)
    	    {
    	    db.x = j;
    	    grid.db = db;
    	    cout << grid << endl;
    	    isOk &= isReductionAddTools_II_Ok(grid);
    	    }
    	}
    return isOk;
    }

/*----------------------------------------------------------------------*\
|*			Device	 					*|
 \*---------------------------------------------------------------------*/

/**
 * protocole :	TID_GLOBAL partout en tabSM
 * resultat:	 (n-1)n/2 ou n=NB_THREAD_GLOBAL
 */
__global__ void fillTidGlobal(int* ptrDevResultGM)
    {
    extern __shared__ int tabSM[];
    reductionIntraThread(tabSM);
    __syncthreads();
    ReductionAddTools::reductionADD(tabSM, ptrDevResultGM);
    }

__device__ void reductionIntraThread(int* tabSM)
    {
    tabSM[threadIdx.x] = blockDim.x * blockIdx.x + threadIdx.x;
    }

/*----------------------------------------------------------------------*\
|*			End	 					*|
 \*---------------------------------------------------------------------*/
