#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include "Device.h"
#include "MonteCarloLong.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration 					*|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void createGenerator(hiprandState* tabDevGeneratorGM, int deviceId);
extern __global__ void monteCarloLong(hiprandState* tabDevGeneratorGM, long nbFleche, long* ptrDevResultat);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur		*|
 \*-------------------------------------*/

MonteCarloLong::MonteCarloLong(const Grid& grid, long nbFlechette, float* ptrPiHat) : ptrPiHat(ptrPiHat), nbFlechette(nbFlechette)
    {
    this->ptrResultat = new long[sizeof(long)];
    this->sizeResGM = sizeof(long);
    this->sizeTabGenerator = grid.threadCounts()*sizeof(hiprandState);

    this->nbFlechetteThread = nbFlechette/(long)grid.threadCounts();
    this->nbFlechette = nbFlechetteThread*(long)grid.threadCounts();
    Device::malloc(&ptrDevResultat, sizeResGM);
    Device::memclear(ptrDevResultat, sizeResGM);
    Device::malloc(&ptrDevtabGenerator, sizeTabGenerator);
    Device::memclear(ptrDevtabGenerator, sizeTabGenerator);

    this->dg = grid.dg;
    this->db = grid.db;
    }

MonteCarloLong::~MonteCarloLong(void)
    {
    Device::free(ptrDevResultat);
    Device::free(ptrDevtabGenerator);
    }


/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void MonteCarloLong::run()
    {
    createGenerator<<<dg, db>>>(ptrDevtabGenerator, 0);
    monteCarloLong<<<dg, db, sizeof(long)*db.x>>>(ptrDevtabGenerator, nbFlechetteThread, ptrDevResultat);
    Device::memcpyDToH(ptrResultat, ptrDevResultat, sizeof(long));
    *ptrPiHat = 4* *ptrResultat / (double)nbFlechette; //Utilisation de m à 4 semble plus précis.
    }

long MonteCarloLong::getFlecheSousCourbe()
    {
    return *ptrResultat;
    }

long MonteCarloLong::getFlecheTiree()
    {
    return nbFlechette;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

