#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include "ReductionAddToolsLock.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

extern __global__ void createGeneratorLong(hiprandState* tabDevGeneratorGM, int deviceId);
__global__ void monteCarloLong(hiprandState* tabDevGeneratorGM, long nbFleche, long* ptrDevResultat);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void intraThread(hiprandState* tabDevGeneratorGM, long nbFleche, long* tabSM);
extern __device__ float fLong(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/



__global__ void monteCarloLong(hiprandState* tabDevGeneratorGM, long nbFleche, long* ptrDevResultat)
    {
    extern __shared__ long tabSM[];
    intraThread(tabDevGeneratorGM, nbFleche, tabSM);
    __syncthreads();
    ReductionAddToolsLock::reductionADD(tabSM, ptrDevResultat);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
__device__ void intraThread(hiprandState* tabDevGeneratorGM, long nbFleche, long* tabSM)
    {
    const int TID = Indice1D::tid();
    hiprandState localGenerator = tabDevGeneratorGM[TID];
    float xAlea;
    float yAlea;
    long nbFlecheIn = 0;
    for (long i = 0; i < nbFleche; ++i)
	{
	xAlea = hiprand_uniform(&localGenerator);
	yAlea = 4 * hiprand_uniform(&localGenerator);

//	    { //Variante 1
//	    if (yAlea < f(xAlea))
//		{
//		nbFlecheIn++;
//		}
//	    }

	    { //Variante 2
	    nbFlecheIn += yAlea < fLong(xAlea);
	    }
	}
    tabSM[Indice1D::tidLocal()] = nbFlecheIn;
    tabDevGeneratorGM[TID] = localGenerator;
    }

__device__ float fLong(float x)
    {
    return 4 / (1 + x * x);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

