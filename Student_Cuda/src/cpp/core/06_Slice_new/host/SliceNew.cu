#include "hip/hip_runtime.h"

#include "SliceNew.h"

#include <iostream>
#include <assert.h>

#include "Device.h"


 /*---------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void kernelSliceSM(float* ptrDevResultat,int nbSlice);

 /*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

 /*-------------------------------------*\
 |*		Constructeur		*|
 \*-------------------------------------*/

SliceNew::SliceNew(const Grid& grid, int nbSlice, float* ptrPiHat) : ptrPiHat(ptrPiHat), nbSlice(nbSlice)
    {
    this->sizeTabGM = sizeof(float); // [octet]

    Device::malloc(&ptrDevResultat, sizeTabGM);
    Device::memclear(ptrDevResultat, sizeTabGM);

    this->dg = grid.dg;
    this->db = grid.db;

    }

SliceNew::~SliceNew(void)
    {
    Device::free (ptrDevResultat);
    }

 /*-------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void SliceNew::run()
    {
    kernelSliceSM<<<dg,db,sizeof(float)*db.x>>>(ptrDevResultat,nbSlice);
    Device::memcpyDToH(ptrPiHat, ptrDevResultat, sizeof(float));
    }

 /*---------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
