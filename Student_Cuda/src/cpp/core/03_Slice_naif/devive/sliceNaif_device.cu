#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void reductionIntraThread(float* tabGM, int nbSlice);
__global__ void ecrasementGM(float* tabGM, int moitier);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ float f(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * Chaque thread effecteur une reduction avec le patern d'entrelacement,
 * puis stocke son résultat dans SA case dans tabGM
 * tabGM est un tableau promu, qui a autant de case que de thread
 */
__global__ void reductionIntraThread(float* tabGM, int nbSlice)
    {
    // TODO SliceNaif
    }

/**
 * Pour version NaivePLus
 *
 * Ramene la semantique du tableau pour une reduction, dans la premiere moitier, par ecraseemt
 * Doit etre appeler plusieurs fois coter host, pour tout ramener dans tabGM[0]
 */
__global__ void ecrasementGM(float* tabGM, int moitier)
    {
    // TODO SliceNaif+
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float f(float x)
    {
    // TODO SliceNaif
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

