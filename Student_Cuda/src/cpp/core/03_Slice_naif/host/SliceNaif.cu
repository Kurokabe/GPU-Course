#include "hip/hip_runtime.h"
#include "SliceNaif.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void reductionIntraThread(float* tabGM,int nbSlice);
extern __global__ void ecrasementGM(float* tabGM, int moitier);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

SliceNaif::SliceNaif(const Grid& grid, int nbSlice, float* ptrPiHat) :
	ptrPiHat(ptrPiHat), nbSlice(nbSlice)
    {
    this->nTabGM = grid.threadCounts();
    this->sizeTabGM = nTabGM * sizeof(float); // [octet]

    // MM
	{
	// TODO SliceNaif
	}

    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}
    }

SliceNaif::~SliceNaif(void)
    {
    //MM (device free)
	{
	// TODO SliceNaif
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void SliceNaif::run()
    {
    // v1 : promotion tab, reduction coter host
	{
	// emploi uniquemnet du kernel : reductionIntraThread

	// TODO SliceNaif

	// Reduction paralle sur cpu du tableau promu ramener coter host
	    {
	    // TODO SliceNaif
	    }
	}

    // v2 : promotion tab, reduction coter device en GM
	{
	// emploi de deux kernels:
	//	reductionIntraThread
	//	ecrasementGM
	//
	// Warning:
	//	(W1) ecrasementGM doit etre lancer dans une boucle coter host
	//	(W2) On serait tenter de realiser cette boucle coter device, c'est techniquement possible, mais pose de gros problème de synchronisation!
	//
	// Note:
	//	Attendez la version de slice en SM pour un code 100% parallel sur GPU, sans boucle sur CPU.


	// TODO SliceNaif+
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
