#include "hip/hip_runtime.h"
#include "MonteCarlo.h"

#include <iostream>
#include <assert.h>
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration 					*|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void createGenerator(hiprandState* tabDevGeneratorGM, int deviceId);
extern __global__ void monteCarlo(hiprandState* tabDevGeneratorGM, int nbFleche, int* ptrDevResultat);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur		*|
 \*-------------------------------------*/

MonteCarlo::MonteCarlo(const Grid& grid, int nbFlechette, float* ptrPiHat) : ptrPiHat(ptrPiHat), nbFlechette(nbFlechette)
    {
    this->ptrResultat = new int[sizeof(int)];
    this->sizeResGM = sizeof(int);
    this->sizeTabGenerator = grid.threadCounts()*sizeof(hiprandState);

    this->nbFlechetteThread = nbFlechette/grid.threadCounts();
    this->nbFlechette = nbFlechetteThread*grid.threadCounts();
    Device::malloc(&ptrDevResultat, sizeResGM);
    Device::memclear(ptrDevResultat, sizeResGM);
    Device::malloc(&ptrDevtabGenerator, sizeTabGenerator);
    Device::memclear(ptrDevtabGenerator, sizeTabGenerator);

    this->dg = grid.dg;
    this->db = grid.db;
    }

MonteCarlo::~MonteCarlo(void)
    {
    Device::free(ptrDevResultat);
    Device::free(ptrDevtabGenerator);
    }


/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void MonteCarlo::run()
    {
    createGenerator<<<dg, db>>>(ptrDevtabGenerator, 0);
    monteCarlo<<<dg, db, sizeof(int)*db.x>>>(ptrDevtabGenerator, nbFlechetteThread, ptrDevResultat);
    Device::memcpyDToH(ptrResultat, ptrDevResultat, sizeof(int));
    *ptrPiHat = 4* *ptrResultat / (double)nbFlechette; //Utilisation de m à 4 semble plus précis.
    }

int MonteCarlo::getFlecheSousCourbe()
    {
    return *ptrResultat;
    }

int MonteCarlo::getFlecheTiree()
    {
    return nbFlechette;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

