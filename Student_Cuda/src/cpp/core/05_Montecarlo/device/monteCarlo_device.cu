#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include "ReductionAddTools.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void createGenerator(hiprandState* tabDevGeneratorGM, int deviceId);
__global__ void monteCarlo(hiprandState* tabDevGeneratorGM, int nbFleche, int* ptrDevResultat);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void intraThread(hiprandState* tabDevGeneratorGM, int nbFleche, int* tabSM);
__device__ float f(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void createGenerator(hiprandState* tabDevGeneratorGM, int deviceId)
    {
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;

    hiprand_init(seed, sequenceNumber, offset, &tabDevGeneratorGM[TID]);
    }

__global__ void monteCarlo(hiprandState* tabDevGeneratorGM, int nbFleche, int* ptrDevResultat)
    {
    extern __shared__ int tabSM[];
    intraThread(tabDevGeneratorGM, nbFleche, tabSM);
    __syncthreads();
    ReductionAddTools::reductionADD(tabSM, ptrDevResultat);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
__device__ void intraThread(hiprandState* tabDevGeneratorGM, int nbFleche, int* tabSM)
    {
    const int TID = Indice1D::tid();
    hiprandState localGenerator = tabDevGeneratorGM[TID];
    float xAlea;
    float yAlea;
    int nbFlecheIn = 0;
    for (int i = 0; i < nbFleche; ++i)
	{
	xAlea = hiprand_uniform(&localGenerator);
	yAlea = 4 * hiprand_uniform(&localGenerator);

//	    { //Variante 1
//	    if (yAlea < f(xAlea))
//		{
//		nbFlecheIn++;
//		}
//	    }

	    { //Variante 2
	    nbFlecheIn += yAlea < f(xAlea);
	    }
	}
    tabSM[Indice1D::tidLocal()] = nbFlecheIn;
    tabDevGeneratorGM[TID] = localGenerator;
    }

__device__ float f(float x)
    {
    return 4 / (1 + x * x);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

