#include "hip/hip_runtime.h"
#include "DamierRGBAFloat.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void damierRGBAFloat(float4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, uint n, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

DamierRGBAFloat::DamierRGBAFloat(const Grid& grid,uint w, uint h, float dt, uint n, const DomaineMath& domaineMath) :
	Animable_I<float4>(grid, w, h, "Damier_Cuda_RGBA_float4", domaineMath), variateurAnimation(Interval<float>(0, 2 * PI), dt)
    {
    // Inputs
    this->n = n;

    // Tools
    this->t = 0;  // protected dans Animable
    }

DamierRGBAFloat::~DamierRGBAFloat()
    {
   // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void DamierRGBAFloat::process(float4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    damierRGBAFloat<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,n,t);
    }

/**
 * Override
 * Call periodicly by the API
 */
void DamierRGBAFloat::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

