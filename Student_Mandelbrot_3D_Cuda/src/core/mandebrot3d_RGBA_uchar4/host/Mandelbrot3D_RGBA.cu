#include "hip/hip_runtime.h"
#include "../../mandebrot3d_RGBA_uchar4/host/Mandelbrot3D_RGBA.h"

#include <assert.h>

#include "Device.h"
#include "MathTools.h"
#include "StringTools.h"
#include "cudaTools.h"

using gpu::DomainMath3D;
using gpu::SurfaceStrip_RGBA_uchar4;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot3DCuda(float3* ptrDevVerticesCoord, uchar4* ptrDevVerticesColor, int w, int h, DomainMath3D domaineMath, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Mandelbrot3D_RGBA::Mandelbrot3D_RGBA(const Grid& grid, int w, int h, const DomainMath3D& domaineMath, float dt, int nMin, int nMax) :
	SurfaceStrip_RGBA_uchar4(w,h,domaineMath ), variateurAnimation(cpu::Interval<float>(nMin, nMax), dt)
    {
    // Inputs/animation
    this->nMin = nMin;
    this->nMax = nMax;
    this->dg = grid.dg;
    this->db = grid.db;

    // Tools
    this->title = "Mandelbrot3D_RGBA_uchar4";
    this->t = nMin;
    }

Mandelbrot3D_RGBA::~Mandelbrot3D_RGBA()
    {
    // rien
    }

/*-------------------------*\
 |*	Surcharge	   *|
 \*------------------------*/

/**
 * Override
 * Call periodicly by the api
 */
void Mandelbrot3D_RGBA::fillVertex(float3* ptrDevVerticesCoord, uchar4* ptrDevVerticesColor, unsigned int w, unsigned int h, const gpu::DomainMath3D& domaineMath)
    {
    mandelbrot3DCuda<<<dg,db>>>(ptrDevVerticesCoord,ptrDevVerticesColor,w,h,domaineMath,t);
    }

/**
 * Override
 * Call periodicly by the api
 */
void Mandelbrot3D_RGBA::animationStep(bool& isNeedUpdate)
    {
    this->t = variateurAnimation.varierAndGet(); // in [nmin,nmax]
    }

/**
 * Override
 * Call periodicly by the api
 */
void Mandelbrot3D_RGBA::paintPrimitives(Graphic2D& graphic2D)
    {
    graphic2D.setFont(TIMES_ROMAN_24);

    float r = 0.5f;
    float g = 0.5f;
    float b = 0.5f;

    graphic2D.setColorRGB(r, g, b);

    // TOP : Para Animation
	{
	string message = "t = " + StringTools::toString(t);
	graphic2D.drawTitleTop(message);
	}

    // BOTTOM : Title
	{
	graphic2D.drawTitleBottom(title);
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

