#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools_GPU.h"
#include "DomainMath3D_GPU.h"
#include "cudaTools.h"
#include "Device.h"

#include "../../mandebrot3d_RGBA_uchar4/device/math/Mandelbrot3DMath_RGBA.h"

using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot3DCuda(float3* ptrDevVerticesCoord, uchar4* ptrDevVerticesColor, int w, int h, DomainMath3D domaineMath, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * w nbPoint en x
 * h nbPoint en y
 */
__global__ void mandelbrot3DCuda(float3* ptrDevVerticesCoord, uchar4* ptrDevVerticesColor, int w, int h, DomainMath3D domaineMath, int n)
    {
    Mandelbrot3DMath_RGBA mandelbrotMath(n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH = w * h;

    uchar4 color;
    float3 sommet;

    float x;
    float y;
    float DX;
    float DY;

    int vertexI;
    int vertexJ;

    domaineMath.delta(w,h,&DX,&DY);

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &vertexI, &vertexJ); // update (vertexI, vertexJ)

	domaineMath.toXY(DX,DY,vertexI,vertexJ,&x,&y);
	float h;
	//mandelbrotMath.sommetXY(&sommet,x,y,t); // update sommet
	mandelbrotMath.colorXY(&color, x, y, &h, n); // update color

	sommet.x = x;
	sommet.y = y;
	sommet.z = mandelbrotMath.z(h);



	ptrDevVerticesColor[s] = color;
	ptrDevVerticesCoord[s] = sommet;

	s += NB_THREAD;
	}

    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

